#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "mini_batch.h"
#include "graph_node.h"


// CUDA核函数 - 将输入元素乘以2
__global__ void multiplyKernel(const double *input, double *output, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        output[idx] = input[idx] * 2.0;
    }
}

void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

// 执行CUDA处理的函数
void runCudaProcess(GraphNode& node, const std::vector<MiniBatch>& inputMiniBatches, std::vector<MiniBatch>& outputMiniBatches, const std::string outputName) {
    // 假设只处理第一个MiniBatch
    if (inputMiniBatches.empty()) return;

    const auto& inputBatch = inputMiniBatches[0].getData();
    int N = inputBatch.size();
    size_t size = N * sizeof(double);

    double *d_input, *d_output;
    checkCudaCall(hipMalloc((void **)&d_input, size));
    checkCudaCall(hipMalloc((void **)&d_output, size));

    // 准备数据
    std::vector<double> h_input(N);
    for (int i = 0; i < N; ++i) {
        h_input[i] = std::get<double>(inputBatch[i]);
    }

    checkCudaCall(hipMemcpy(d_input, h_input.data(), size, hipMemcpyHostToDevice));

    // 计算grid和block大小
    int block_size = 128;
    int grid_size = (N + block_size - 1) / block_size;

    // 调用CUDA核函数
    multiplyKernel<<<grid_size, block_size>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    // 从GPU内存复制回主机内存
    std::vector<double> h_output(N);
    checkCudaCall(hipMemcpy(h_output.data(), d_output, size, hipMemcpyDeviceToHost));

    // 准备输出MiniBatch
    outputMiniBatches.clear();
    std::vector<DataContainer> outputData(N);
    for (int i = 0; i < N; ++i) {
        outputData[i] = h_output[i];
    }
    outputMiniBatches.emplace_back(outputName, outputData);

    hipFree(d_input);
    hipFree(d_output);
}