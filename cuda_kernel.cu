#include <hip/hip_runtime.h>
#include <iostream>

// CUDA核函数 - 向量加法
__global__ void addKernel(const double *x, const double *y, double *z, int N) {
    int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N) {
        z[n] = x[n] + y[n];
    }
}

// 检查CUDA调用的结果
void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

// 在GPU上执行向量加法的函数
void runCudaAddKernel(const double *h_x, const double *h_y, double *h_z, int N) {
    size_t size = N * sizeof(double);
    
    double *d_x, *d_y, *d_z;
    checkCudaCall(hipMalloc((void **)&d_x, size));
    checkCudaCall(hipMalloc((void **)&d_y, size));
    checkCudaCall(hipMalloc((void **)&d_z, size));

    checkCudaCall(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice));

    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;

    addKernel<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    hipDeviceSynchronize(); // 等待CUDA核函数完成

    checkCudaCall(hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost));

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}
