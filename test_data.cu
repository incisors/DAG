#include <iostream>
#include <numeric> // For std::accumulate

#include "graph_node.h"

void sumIntegersCPU() {
    std::vector<int> data = {1, 2, 3, 4, 5}; // 示例数据
    int sum = std::accumulate(data.begin(), data.end(), 0);

    std::cout << "Sum (CPU): " << sum << std::endl;
}

int main() {
    // 创建一个 CPU 类型的 GraphNode 实例
    GraphNode node(ComputeType::CPU);

    // 设置 CPU 处理函数
    node.
    node.setCPUProcess(sumIntegersCPU);

    // 执行节点
    node.execute();

    return 0;
}